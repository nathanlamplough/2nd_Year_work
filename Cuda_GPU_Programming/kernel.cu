#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <random>
#include <string>
#include <unordered_map>
#include <algorithm>
#include <vector>
#include "windows.h"
#include "psapi.h"
#include <math.h> 

#define OTL_ODBC 
#define OTL_ODBC_SELECT_STM_EXECUTE_BEFORE_DESCRIBE
#include <otlv4.h>

using namespace std;


int IDs[1470908];
otl_connect db;
mt19937 random_generator;
const int vector_size = 9777;
const int n_vectors = 100;

int single_vector[vector_size];
float * vectors = new float[vector_size * n_vectors];

//class to help store cluster and distance data for vectors
class c_data
{
public:

	int cluster;
	float distance;
};

//dictionary to store the vector and its cluster and distance from centroid 
unordered_map<int, c_data> cluster_data;

void print_memory_usage() {
	PROCESS_MEMORY_COUNTERS pmc;
	GetProcessMemoryInfo(GetCurrentProcess(), &pmc, sizeof(pmc));
	SIZE_T virtualMemUsedByMe = pmc.WorkingSetSize;
	cout << "************ Current Memory Usage: " << virtualMemUsedByMe / 1048576 << " Megabytes ************" << endl;
}

void print_cluster_data() {
	ofstream myfile;
	myfile.open("C:/Users/natha/source/repos/Kmeans_cuda2/Kmeans_cuda2/data/cluster_data.txt");
	for (int i = 0; i < 1470908; i++) {
		int id = IDs[i];
		myfile << id << " " << cluster_data[id].cluster << " " << cluster_data[id].distance << endl;
	}
}

//computes the distance by column from centroid to vector and squares to avoid negative numbers
__global__ void dist(float * dist_out, float * d_vect, float * d_cent) {
	int row = threadIdx.x;
	int column = blockIdx.x;
	float distance = d_cent[column] - d_vect[row * vector_size + column];
	float sq_distance = distance * distance;
	dist_out[row * vector_size + column] = sq_distance;
}

//sums the column distances to give a scalar distance
__global__ void sumDist(float * d_dist_out, float * d_dist_in) {
	float sum = 0;
	int row = threadIdx.x;
	for (int i = 0; i < vector_size; i++) {
		sum = sum + d_dist_in[vector_size * row + i];
	}
	d_dist_out[row] = sum;
}
//computes the mean of each column over all vectors assigned to each cluster, the mean is the columns is then the new centroid column value
__global__ void mean(float * d_cent_out, int * d_vect_in, int * d_ends, int * d_cent_in) {
	float mean = 0;
	float sum = 0;
	int cluster = threadIdx.x;
	int column = blockIdx.x;

	for (int i = 0; i < d_ends[cluster]; i++) {
		sum = sum + d_vect_in[d_cent_in[cluster * n_vectors + i] * vector_size + column];
	}
	mean = sum / d_ends[cluster];
	d_cent_out[column] = mean;
}


//functions to connect to SQL database and retrieve word vectors for clustering into RAM
class database_functions {

public:

	void select_vectors() {
		try {

			db.rlogon("natha/@Vectors");

			int c = 0;
			for (int j = 0; j < n_vectors; j++) {
				otl_long_string Vector(40000);
				db.set_max_long_size(40000);
				otl_connect::otl_initialize();

				otl_stream i(50,
					"SELECT * FROM WordVectors "
					"where ID=:ID<int>",

					db
				);


				int ID;
				int id = IDs[j];
				i << id;


				while (!i.eof()) {
					i >> ID;
					i >> Vector;
					string current_value = "";

					for (int j = 0; Vector[j] != 0; ++j) {
						int converted_char = Vector[j];
						if (converted_char == 32) {
							int int_value = stoi(current_value);
							vectors[c] = int_value;
							current_value = "";
							c++;
						}
						else {
							int real_value = converted_char - 48;
							string string_value = to_string(real_value);
							current_value += string_value;
						}
					}
					int int_value = stoi(current_value);
					vectors[c] = int_value;
					c++;

				}

			}
		}
		catch (otl_exception& p) {
			cerr << p.msg << endl;
			cerr << p.stm_text << endl;
			cerr << p.var_info << endl;
		}

		db.logoff();
	}

	void select_one_vector(int id) {
		otl_connect::otl_initialize();

		try {

			db.rlogon("natha/@Vectors");

			otl_long_string Vector(40000);
			db.set_max_long_size(40000);

			otl_stream i(50,
				"SELECT * FROM WordVectors "
				"where ID=:ID<int>",

				db
			);


			int ID;
			i << IDs[id];

			while (!i.eof()) {
				i >> ID;
				i >> Vector;
				string current_value = "";
				int c = 0;
				for (int j = 0; Vector[j] != 0; ++j) {
					int converted_char = Vector[j];
					if (converted_char == 32) {
						int int_value = stoi(current_value);
						single_vector[c] = int_value;
						current_value = "";
						c++;
					}
					else {
						int real_value = converted_char - 48;
						string string_value = to_string(real_value);
						current_value += string_value;
					}
				}
				int int_value = stoi(current_value);
				single_vector[c] = int_value;

			}
		}

		catch (otl_exception& p) {
			cerr << p.msg << endl;
			cerr << p.stm_text << endl;
			cerr << p.var_info << endl;
		}

		db.logoff();
	}


	void select_ids() {
		otl_stream i(50, // buffer size
			"SELECT ID FROM WordVectors",
			db
		);

		int ID;

		int count = 0;
		while (!i.eof()) {
			i >> ID;
			IDs[count] = ID;
			count++;

		}
	}
	void intialise_database() {

		otl_connect::otl_initialize();
		try {

			db.rlogon("natha/@Vectors");
			print_memory_usage();
			select_ids();


		}
		catch (otl_exception& p) {
			cerr << p.msg << endl;
			cerr << p.stm_text << endl;
			cerr << p.var_info << endl;
		}

		db.logoff();


	}
};




hipError_t kmeans(database_functions dbc)
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);


	const int clusters = 20;
	const int iterations = 10;
	//int vectors[1000 * 100];
	int centroidsIndex[clusters];
	float * centroids = new float[clusters * vector_size];

	//random_generator.seed(5);
	//uniform_int<int> rand25(1, 25);
	uniform_int<int> rand1000(0, 1470908);
	////randomly create vectors for testing purposes
	//for (int i = 0; i < 1000 * 100; i++) {
	//	vectors[i] = rand25(random_generator);
	//}
	//randomly assign centroids
	for (int c = 0; c < clusters; c++) {
		centroidsIndex[c] = rand1000(random_generator);
	}

	//create copies of centroids 
	int row = 0;
	for (const auto& index : centroidsIndex) {
		dbc.select_one_vector(index);
		for (int c = 0; c < vector_size; c++) {
			centroids[row * vector_size + c] = single_vector[c];
		}
		row++;
	}

	bool first = true;

	//get the sizes the variables for memcpy
	const int ARRAY_BYTES_VECTORS = sizeof(float) * n_vectors * vector_size;
	const int ARRAY_BYTES_CENTROIDS = sizeof(float) * clusters * vector_size;
	const int ARRAY_BYTES_DISTANCES = sizeof(float) * n_vectors * vector_size;

	//run kmeans for specified iterations
	for (int iter = 0; iter < iterations; iter++) {
		//loop and find distances to vectors for each centroid
		for (int vc = 0; vc < 150; vc++) {
			dbc.select_vectors();

			cout << "------------ Beginning round " << vc + 1 << " of " << n_vectors << " vectors" << " ------------" << endl;
			print_memory_usage();
			if (iter < 1) {
				first = true;
			}
			else {
				first = false;
			}
			for (int fv = 0; fv < n_vectors; fv++) {
				c_data c;
				cluster_data.insert({ IDs[n_vectors * vc + fv], c });
			}

			for (int cent = 0; cent < clusters; cent++) {
				float centroid[vector_size];
				//copy centroid to 1D array so can easily copy to kernel
				for (int i = 0; i < vector_size; i++) {
					centroid[i] = centroids[cent * vector_size + i];
				}

				//kernel variables: vectors, distances output, centroids
				float * d_vect;
				float * dist_out;
				float * d_cent;
				//storage for outputted distances

				float * ind_distances = new float[n_vectors * vector_size];
				//allocating memory
				hipMalloc((void**)&d_vect, ARRAY_BYTES_VECTORS);
				hipMalloc((void**)&d_cent, ARRAY_BYTES_CENTROIDS / clusters);
				hipMalloc((void**)&dist_out, ARRAY_BYTES_DISTANCES);

				//copying variables to gpu
				cudaStatus = hipMemcpy(d_vect, vectors, ARRAY_BYTES_VECTORS, hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "vectors memcpy to device failed");
					return cudaStatus;
				}

				cudaStatus = hipMemcpy(d_cent, centroid, ARRAY_BYTES_CENTROIDS / clusters, hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "centroid memcpy to device failed");
					return cudaStatus;
				}

				//intiating kernel and computing distances per column, 1 block for each vector and 1 thread for each dimension
				dist << <vector_size, n_vectors >> > (dist_out, d_vect, d_cent);

				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "distance kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
					return cudaStatus;
				}
				//copying distances to cpu

				cudaStatus = hipMemcpy(ind_distances, dist_out, ARRAY_BYTES_VECTORS, hipMemcpyDeviceToHost);

				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "vectors memcpy to host failed");
					return cudaStatus;
				}

				hipFree(d_vect);
				hipFree(dist_out);
				hipFree(d_cent);

				//kernel variables: summed distances output, column distances input
				float * d_dist_out;
				float * d_dist_in;
				//storage for summed distances
				float distances[n_vectors];

				//allocating memory
				hipMalloc((void**)&d_dist_out, ARRAY_BYTES_DISTANCES / vector_size);
				hipMalloc((void**)&d_dist_in, ARRAY_BYTES_VECTORS);

				cudaStatus = hipMemcpy(d_dist_in, ind_distances, ARRAY_BYTES_VECTORS, hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "individual distances memcpy to device failed");
					return cudaStatus;
				}

				//summing the distances of individual columns, doing 1 vector per thread
				sumDist << <1, n_vectors >> > (d_dist_out, d_dist_in);
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "summing distance kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
					return cudaStatus;
				}

				cudaStatus = hipMemcpy(distances, d_dist_out, ARRAY_BYTES_VECTORS / vector_size, hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "summed distances memcpy to host failed");
					return cudaStatus;
				}

				hipFree(d_dist_out);
				hipFree(d_dist_in);

				//if this is the first centroid and iteration then populating the dictionary with values
				if (first == true) {
					for (int d = 0; d < n_vectors; d++) {
						c_data data;
						data.cluster = 0;
						data.distance = distances[d];
						int id = IDs[n_vectors * vc + d];
						cluster_data[id] = data;
					}
					first = false;
				}

				//updating the dictionary if distances are closer
				else {
					for (int d = 0; d < n_vectors; d++) {
						if (distances[d] < cluster_data[IDs[n_vectors * vc + d]].distance) {
							int id = IDs[n_vectors * vc + d];
							cluster_data[id].distance = distances[d];
							cluster_data[id].cluster = cent;
						}
					}
				}

				free(ind_distances);
				cout << cent + 1 << " centroids processed" << endl;
			}

			//filling a flattened 2D array with vectors IDs/indexes grouped with their cluster
			//each row is length n_vectors batch and represents a different cluster
			int assignedClusters[clusters * n_vectors];
			int ends[clusters];
			std::fill_n(assignedClusters, clusters * n_vectors, -1);
			std::fill_n(ends, clusters, 0);
			for (int d = 0; d < n_vectors; d++) {
				int clust = cluster_data[IDs[n_vectors * vc + d]].cluster;
				assignedClusters[n_vectors * clust + ends[clust]] = d;
				ends[clust]++;
			}
			//kernel variables: updated centroids output, vectors, the number of vectors assigned to each cluster
			float * d_cent_out;
			int * d_vect_in;
			int * d_ends;
			int * d_cent_in;

			//allocating memory
			hipMalloc((void**)&d_cent_out, ARRAY_BYTES_CENTROIDS / clusters);
			hipMalloc((void**)&d_vect_in, ARRAY_BYTES_VECTORS);
			hipMalloc((void**)&d_ends, sizeof(int) * clusters);
			hipMalloc((void**)&d_cent_in, (ARRAY_BYTES_VECTORS / vector_size) * clusters);

			//copying variables to gpu
			hipError_t cudaStatus = hipMemcpy(d_cent_in, assignedClusters, (ARRAY_BYTES_VECTORS / vector_size) * clusters, hipMemcpyHostToDevice);

			if (cudaStatus != hipSuccess) {

				fprintf(stderr, "assigned centroids hipMemcpy to device failed!");
				return cudaStatus;
			}

			hipError_t cudaStatus = hipMemcpy(d_vect_in, vectors, ARRAY_BYTES_VECTORS, hipMemcpyHostToDevice);

			if (cudaStatus != hipSuccess) {

				fprintf(stderr, "vectors hipMemcpy to device failed!");
				return cudaStatus;
			}

			hipError_t cudaStatus = hipMemcpy(d_ends, ends, sizeof(int) * clusters, hipMemcpyHostToDevice);

			if (cudaStatus != hipSuccess) {

				fprintf(stderr, "index ends of clusters hipMemcpy to host failed!");
				return cudaStatus;
			}

			//computing the mean of each of the 100 columns per centroids/cluster
			mean << <vector_size, clusters >> > (d_cent_out, d_vect_in, d_ends, d_cent_in);

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "computing cluster means kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				return cudaStatus;
			}

			//returning the new updated centroids
			hipError_t cudaStatus = hipMemcpy(centroids, d_cent_out, ARRAY_BYTES_CENTROIDS / clusters, hipMemcpyDeviceToHost);

			//checking if copy is sucessful
			if (cudaStatus != hipSuccess) {

				fprintf(stderr, "updated centroids hipMemcpy to host failed!");
				return cudaStatus;
			}


			hipFree(d_vect_in);
			hipFree(d_ends);
			hipFree(d_cent_in);
			hipFree(d_cent_out);
			//free(vectors);
		}

	}
	return cudaStatus;
}
//2283mb per 100k

int main() {
	database_functions db;
	db.intialise_database();
	cout << "IDs retrieved from database" << endl;
	kmeans(db);
	return 0;
}







